#include "hip/hip_runtime.h"
#include "erl_nif.h"

__global__
void dot_product(float *ref4, float *a, float *b, int n)
{
__shared__ float cache[256];
	int tid = (threadIdx.x + (blockIdx.x * blockDim.x));
	int cacheIndex = threadIdx.x;
	float temp = 0.0;
while((tid < n)){
	temp = ((a[tid] * b[tid]) + temp);
	tid = ((blockDim.x * gridDim.x) + tid);
}
	cache[cacheIndex] = temp;
__syncthreads();
	int i = (blockDim.x / 2);
while((i != 0)){
if((cacheIndex < i))
{
	cache[cacheIndex] = (cache[(cacheIndex + i)] + cache[cacheIndex]);
}

__syncthreads();
	i = (i / 2);
}
if((cacheIndex == 0))
{
	ref4[blockIdx.x] = cache[0];
}

}

extern "C" void dot_product_call(ErlNifEnv *env, const ERL_NIF_TERM argv[], ErlNifResourceType* type)
  {

    ERL_NIF_TERM list;
    ERL_NIF_TERM head;
    ERL_NIF_TERM tail;
    float **array_res;

    const ERL_NIF_TERM *tuple_blocks;
    const ERL_NIF_TERM *tuple_threads;
    int arity;

    if (!enif_get_tuple(env, argv[1], &arity, &tuple_blocks)) {
      printf ("spawn: blocks argument is not a tuple");
    }

    if (!enif_get_tuple(env, argv[2], &arity, &tuple_threads)) {
      printf ("spawn:threads argument is not a tuple");
    }
    int b1,b2,b3,t1,t2,t3;

    enif_get_int(env,tuple_blocks[0],&b1);
    enif_get_int(env,tuple_blocks[1],&b2);
    enif_get_int(env,tuple_blocks[2],&b3);
    enif_get_int(env,tuple_threads[0],&t1);
    enif_get_int(env,tuple_threads[1],&t2);
    enif_get_int(env,tuple_threads[2],&t3);

    dim3 blocks(b1,b2,b3);
    dim3 threads(t1,t2,t3);

    list= argv[3];

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg1 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg2 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg3 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  int arg4;
  enif_get_int(env, head, &arg4);
  list = tail;

   dot_product<<<blocks, threads>>>(arg1,arg2,arg3,arg4);
    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
     }
}
