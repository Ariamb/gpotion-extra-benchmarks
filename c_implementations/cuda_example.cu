
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < 10) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[10], b[10], c[10];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < 10; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMalloc((void **)&d_a, 10 * sizeof(int));
    hipMalloc((void **)&d_b, 10 * sizeof(int));
    hipMalloc((void **)&d_c, 10 * sizeof(int));

    hipMemcpy(d_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 10 * sizeof(int), hipMemcpyHostToDevice);

    add<<<10, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, 10 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
